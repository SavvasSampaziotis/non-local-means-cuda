#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "reduction.h"


// void init_reduction_cache(int rowLength, int rowNum, int threads_num, /*ouit*/ ReductionCache* rc)
// {
// 	rc->blockDim.x = threads_num;
// 	rc->blockDim.y = 1;
// 	rc->blockDim.z = 1;

// 	int blocks_num = ceil(rowLength/threads_num); 
// 	if(blocks_num==0) blocks_num=1;

// 	rc->blocksNum = blocks_num;

// 	rc->gridDim.x = blocks_num;
// 	rc->gridDim.y = rowNum; // One row of block for each matrix row 
// 	rc->gridDim.z = 1;

// 	rc->rowNum = rowNum;
// 	rc->reduced_vec_length = rowNum*blocks_num; // ronNum * (number of blocks per row) 
	
// 	rc->cache_size = rowNum*threads_num*sizeof(float);
// 	if(rc->cache_size > 1024*16) // cache > 16 KB. CUCA 1.x allows max sm 16 per MP
// 		printf("[WARNING]:\t[INIT_REDUCTION_CACHE]:\t \
// 			Shared Memory size too large: %lu\n",\ rc->cache_size);

// 	if(blocks_num>1)  
// 		hipMalloc((void**) &(rc->d_reduced_vec), rc->reduced_vec_length*sizeof(float));
// 		// This is not needed in this case. As reduction cache, d_sum can also be used.

// 	hipMalloc((void**) &(rc->d_sum), rowNum*sizeof(float));
// }


// void delete_reduction_cache(ReductionCache* reductionCache)
// {
// 	if(reductionCache->blocksNum>1)
// 		hipFree(reductionCache->d_reduced_vec);
// 	hipFree(reductionCache->d_sum);
// }


// void WR_reduction(int N, float* d_A, /*out*/ ReductionCache* rc )
// {
// 	if(rc->blocksNum == 1)
// 	{
// 		// We need only one reduction call!
// 		reduction_sum <<<rc->gridDim, rc->blockDim, rc->cache_size>>>(N, d_A, rc->d_sum);

// 		//no need for the d_reduction cache 			
// 	}
// 	else
// 	{	
// 		// We need multiple reduction calls!
// 		reduction_sum <<<rc->gridDim, rc->blockDim, rc->cache_size>>>(N, d_A, rc->d_reduced_vec);		
			
// 		/* Reduct the final reduction vector! */
	
// 		/* Ideally we would like threads_num==length(reduced_vec)/numRow. 
// 		However threads_num2 must be a power of 2. Thus:
// 		*/
// 		int threads_num2 = exp2f(floor(log2f(rc->reduced_vec_length/rc->rowNum))); 
// 		if(threads_num2>512)
// 			threads_num2=512;
// 		//printf("THREADS: %d RED_VEC %d\n", threads_num2, rc->reduced_vec_length/rc->rowNum );

// 		dim3 gridDim2(1,rc->rowNum,1);
// 		dim3 blockDim2(threads_num2,1,1);
// 		reduction_sum<<<gridDim2, blockDim2, threads_num2*sizeof(float)>>>\
// 			(rc->gridDim.x, rc->d_reduced_vec, rc->d_sum); //

// 		// WARNING: launching with original thread_num might be too much. 
// 		// SOLUTION: Find power-of-2 nearest to block_num 
// 	}	
// }


__device__
float reduction_op(float a, float b, int op)
{
	switch(op)
	{
		case R_SUM:
			return a+b;
		case R_MAX:
			return (a>b) ? a:b;
		default:
			return a+b;
	}
}


/*
	This is the reduction function used for a variety of matrix operations. 

*/
__device__
void reduction(int N, float* X, float* reducted_vec, int op)
{
	extern __shared__ float reduction_cache[] ;

	//thread ID on each row of blocks
	int tid = blockDim.x * blockIdx.x + threadIdx.x; 
	int cache_i = threadIdx.x;

	int tid_y = blockIdx.y;

	/* This UNROLLS the elements of x, "outside" the grid's index range.
		In the case of N=600, threadsPerBlock=256 and 2 blocks in total, 
		we have 600-256*2=88 additions done in parallel, before the reduction of the 512 threads.

		incase the index-range > N, the reduction scheme will simply add some zeros to the vector. 
		This allows as to oversubscribe in terms of threads and blocks. 
	*/
	int offset = N*tid_y;
	float temp = 0; // reduction_op_init
	while (tid < N)
	{
		temp = reduction_op(temp, X[tid+offset], op); 
		tid += blockDim.x * gridDim.x;
	}

	/* Load x-data  into local shared memory. 
		As mentioned before, some entries are small sums of
		 x's outside the grid's range  */
	reduction_cache[cache_i] = temp;	
	__syncthreads();
	
	// Begin the reduction per shared-memory-block
	for(int i=blockDim.x/2; i>0; i>>=1)
	{	
		if(cache_i < i)
			reduction_cache[cache_i] = reduction_op(reduction_cache[cache_i], reduction_cache[cache_i+i], op);  
		__syncthreads();
	}

	// Unroll Last warp
	/*if(cache_i>32)
	{
		reduction_cache[cache_i] += reduction_cache[cache_i+32];
		reduction_cache[cache_i] += reduction_cache[cache_i+16];
		reduction_cache[cache_i] += reduction_cache[cache_i+8];
		reduction_cache[cache_i] += reduction_cache[cache_i+4];
		reduction_cache[cache_i] += reduction_cache[cache_i+2];
		reduction_cache[cache_i] += reduction_cache[cache_i+1];  
	}*/

	// Final Sum is stored in global array.
	if(cache_i==0)
		reducted_vec[blockIdx.y*gridDim.x + blockIdx.x] = reduction_cache[0];
}

__global__
void rowsum(int N, float* X, float* reducted_vec)
{
	reduction(N,X,reducted_vec, R_SUM);
}


__global__
void rowmax(int N, float* X, float* reducted_vec)
{
	reduction(N,X,reducted_vec, R_MAX);
}