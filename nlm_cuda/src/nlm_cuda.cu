#include "hip/hip_runtime.h"



/*
	Source: https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
*/
__device__
int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

__device__ 
int getGlobalIdx_2D_3D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) \
					+ (threadIdx.z * (blockDim.x * blockDim.y)) 	\
					+ (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}


__global__
void generate_3D_cube(float *d_image, float *d_patchCube, int H, int W, int pH, int pW)
{
	int k = getGlobalIdx_2D_2D();	
	
	// Local coordinates on patch window (ref pixel at the center)
	int patch_i = threadIdx.y - (pH-1)/2;
	int patch_j = threadIdx.x - (pW-1)/2;
	
	int im_i = blockIdx.y;
	int im_j = blockIdx.x;

	int i = im_i+patch_i;
	int j = im_j+patch_j;

	if ( (i>=0) && (j>=0) && (i<H) && (j<W) )
		d_patchCube[k] = d_image[ i*W + j ];
	else
		d_patchCube[k] = 0; // this is the zero-padding occuring in array boundaries...

}

__device__
float gaussian2D(float x, float y, float s_x, float s_y)
{
	float a = x*x/(s_x*s_x);
	float b = y*y/(s_y*s_y);

	return expf( -(a+b)/2 );
}



__global__
void apply_gaussian_filter(float *d_patchCube, int pH, int pW, float patchSigma_h, float patchSigma_w)
{	
	// Local coordinates on patch window (ref pixel at the center)
	int patch_i = threadIdx.y - (pH-1)/2;
	int patch_j = threadIdx.x - (pW-1)/2;

	// Calc Gaussian Filter value on patch coordinates
	float gaussCoeff = gaussian2D(patch_i, patch_j, patchSigma_h, patchSigma_w);

	// Multiply by existing patch-element
	int k = getGlobalIdx_2D_2D();
	d_patchCube[k] = gaussCoeff*d_patchCube[k];
}


__global__
void calc_dist_matrix(float *d_distMatrix, float *d_patchCube, float sigma)
{
	int i = blockIdx.y*gridDim.y + blockIdx.y;
	int j = blockIdx.x*gridDim.x + blockIdx.X;


	d_distMatrix[k] = d_patchCube

}