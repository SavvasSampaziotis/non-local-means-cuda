

#include "array_utilities.h"
#include <stdio.h>
#include <stdlib.h>

#include "nlm_cuda.h"


int H,W;
float *data;

void test_generate_3d_cube();
void test_apply_gaussianfilt();


int main(int argc, char** argv)
{
	
	char* filename;
	if(argc == 2)
		filename = argv[1];
	else
		printf("NO ARGS GIVEN\n");
	printf("[MAIN]:\t.dat File targeted for denoising: %s", filename);
	read_dataset(&H, &W, &data, filename);
	printf("%f\n\n",data[3]);


	/***** Test Area *****/

	// test_generate_3d_cube();	

	test_apply_gaussianfilt();
	// 

	// Free data
	free(data);
	return 0;
}




void test_generate_3d_cube(){
	
	printf("Generate 3D Cube\n" );

	float* d_dist; // Memory Container of the 2D H-by-W image (in 1-by-HW 1D array)
	float* d_patchCube; // Memory Container of the 3D H-by-W-by-patchSize cube containing the (in 1-by-HW 1D array)

	int H = 4, W = 5;
	int N = H*W;
	int pH = 3, pW=3; 
	int M = pH*pW;

	float* testArray = (float*) malloc(N*sizeof(float));
	for(int i=0; i<N*M; i++)
		testArray[i] = i+1;

	hipMalloc( (void**) &d_patchCube, M*N*sizeof(float) );
	hipMemcpy(d_patchCube, testArray, N*sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc( (void**) &d_dist, N*sizeof(float) );
	
	//////////////
	dim3 blockDim2D	( pW, pH, 1 ); 
  	dim3 gridDim2D	( W, H, 1 ); 
	calc_dist_matrix<<< gridDim2D, blockDim2D >>> (d_dist, d_patchCube, H,W, pH,pW);


	////////////// Print Results	
	printf("Test Array:\n");
	print_array(H,W,testArray);
	
	float* dist = (float*) malloc(N*N*sizeof(float)) ;	
	hipMemcpy(dist, d_dist, N*N*sizeof(float), hipMemcpyDeviceToHost);
	
	if (M < 10)
	{
		printf("pHxpW Patches:\n");
		print_array(H*W, pH*pW, patchCube);
	}
	else
		printf("Array too big to print in console\n");

	//// Clean Up
	hipFree(d_dist);
	hipFree(d_patchCube);
	free(dist);
}




void test_apply_gaussianfilt(){
	
	printf("Apply Gaussian Filter on each patch of the 3D Cube\n" );

	float* d_patchCube; // Memory Container of the 3D H-by-W-by-patchSize cube containing the (in 1-by-HW 1D array)

	float patchSigma = 1;
	int pH = 7, pW = 5; 
	
	int M = pH*pW;
	int H = 4, W = M;
	int N = H*W;

	float* testArray = (float*) malloc(M*N*sizeof(float));
	for(int i=0; i<M*N; i++)
		testArray[i] = 1;

	hipMalloc( (void**) &d_patchCube, M*N*sizeof(float) );
	hipMemcpy(d_patchCube, testArray, M*N*sizeof(float), hipMemcpyHostToDevice);
	
	
	//////////////
	dim3 blockDim2D	( pW, pH, 1 ); 
  	dim3 gridDim2D	( W, H, 1 ); 
	
	apply_gaussian_filter<<< gridDim2D, blockDim2D >>>( d_patchCube,  pH, pW,  1, 2);


	////////////// Print Results
	
	printf("\nTest Array:");
	print_array(N,M,testArray);

	float* patchCube = (float*) malloc(M*N*sizeof(float)) ;	
	hipMemcpy(patchCube, d_patchCube, M*N*sizeof(float), hipMemcpyDeviceToHost);
	
	
	printf("\npHxpW Patches:");
	print_array(3, M, patchCube);


	//// Clean Up
	hipFree(d_patchCube);
	free(patchCube);
}



void test_generate_3d_cube(){
	
	printf("Generate 3D Cube\n" );


	float* d_image; // Memory Container of the 2D H-by-W image (in 1-by-HW 1D array)
	float* d_patchCube; // Memory Container of the 3D H-by-W-by-patchSize cube containing the (in 1-by-HW 1D array)

	int H = 4, W = 5;
	int N = H*W;
	int pH = 3, pW=3; 
	int M = pH*pW;

	float* testArray = (float*) malloc(N*sizeof(float));
	for(int i=0; i<N; i++)
		testArray[i] = i+1;

	hipMalloc( (void**) &d_image, N*sizeof(float) );
	hipMalloc( (void**) &d_patchCube, M*N*sizeof(float) );
	hipMemcpy(d_image, testArray, N*sizeof(float), hipMemcpyHostToDevice);
	
	
	//////////////
	dim3 blockDim2D	( pW, pH, 1 ); 
  	dim3 gridDim2D	( W, H, 1 ); 
	generate_3D_cube<<< gridDim2D, blockDim2D >>> (d_image, d_patchCube, H,W, pH,pW);


	////////////// Print Results
	if (W < 10)
	{
		printf("Test Array:\n");
		print_array(H,W,testArray);
	}
	else
		printf("Array too big to print in console\n");

	float* patchCube = (float*) malloc(M*N*sizeof(float)) ;	
	hipMemcpy(patchCube, d_patchCube, M*N*sizeof(float), hipMemcpyDeviceToHost);
	
	if (M < 10)
	{
		printf("pHxpW Patches:\n");
		print_array(H*W, pH*pW, patchCube);
	}
	else
		printf("Array too big to print in console\n");

	//// Clean Up
	hipFree(d_image);
	hipFree(d_patchCube);
	free(patchCube);
}